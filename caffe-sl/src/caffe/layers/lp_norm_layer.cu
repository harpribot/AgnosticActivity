#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/lp_norm_layer.hpp"

namespace caffe {

template <typename Dtype>
void L2NormLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = top[0]->count();
  const int num = top[0]->shape(0);
  const int dim = count / num;

  Dtype * inv_norm = this->inv_lpnorm_;

  for (int i=0; i<num; ++i) {
    const Dtype * bottom_data = bottom[0]->gpu_data() + bottom[0]->offset(i);
    caffe_gpu_dot(dim, bottom_data, bottom_data, inv_norm + i);
  }
  for (int i=0; i<num; ++i) {
    const Dtype * bottom_data = bottom[0]->gpu_data() + bottom[0]->offset(i);
    Dtype * top_data = top[0]->mutable_gpu_data() + top[0]->offset(i);
    inv_norm[i] = 1.0 / std::sqrt(inv_norm[i]);
    caffe_gpu_scale(dim, inv_norm[i], bottom_data, top_data);
  }
}

template <typename Dtype>
void L2NormLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }

  const int count = top[0]->count();
  const int num = top[0]->shape(0);
  const int dim = count / num;

  Dtype * inv_norm = this->inv_lpnorm_;
  for (int i=0; i<num; ++i) {
    const Dtype* top_data = top[0]->gpu_data() + top[0]->offset(i);
    Dtype* top_diff = top[0]->mutable_gpu_diff() + top[0]->offset(i);
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff() + bottom[0]->offset(i);

    Dtype temp;
    caffe_gpu_scale(dim, inv_norm[i], top_diff, top_diff);
    caffe_gpu_dot(dim, top_data, top_diff, &temp);
    caffe_gpu_scale(dim, -temp, top_data, bottom_diff);
    caffe_gpu_add(dim, bottom_diff, top_diff, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(L2NormLayer);


}  // namespace caffe
